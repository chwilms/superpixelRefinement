#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/normalized_sigmoid_cross_entropy_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SigmoidCrossEntropyLossIgnoreDiffGPU(const int count,
    const int ignore_label, const Dtype* target, Dtype* diff) {
  CUDA_KERNEL_LOOP(i, count) {
    const int target_value = static_cast<int>(target[i]);
    if (target_value == ignore_label) {
      diff[i] = 0;
    }
  }
}


template <typename Dtype>
void NormalizedSigmoidCrossEntropyLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) { LOG(FATAL) << this->type() << " Layer cannot backpropagate to label inputs."; }
  if (propagate_down[0]) {
    // First, compute the diff
    const int count = bottom[0]->count();
    const int num = bottom[0]->num();
    const int width = bottom[0]->shape(2); const int height = bottom[0]->shape(3);
    const int factor = bottom[0]->shape(1) * bottom[0]->shape(2) * bottom[0]->shape(3);

    const Dtype* sigmoid_output_data = sigmoid_output_->gpu_data();
    const Dtype* target = bottom[1]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    caffe_copy(count, sigmoid_output_data, bottom_diff);
    caffe_gpu_axpy(count, Dtype(-1), target, bottom_diff);
    // Zero out gradient of ignored targets.
    if (has_ignore_label_) {
      // NOLINT_NEXT_LINE(whitespace/operators)
      SigmoidCrossEntropyLossIgnoreDiffGPU<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(count, ignore_label_, target, bottom_diff);
    }
    // Scale down gradient
    const Dtype loss_weight = top[0]->cpu_diff()[0];
    caffe_gpu_scal(count, loss_weight / ( num * width * height ), bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_BACKWARD(NormalizedSigmoidCrossEntropyLossLayer);


}  // namespace caffe
