#include "hip/hip_runtime.h"
#include "caffe/layers/sliding_window_layer.hpp"
#include <cstdio>


namespace caffe {

template <typename Dtype>
__global__ void SlidingWindowForward(const int nthreads, const Dtype* bottom_data,
    const int channels, const int bottom_height, const int bottom_width, 
    const int top_height, const int top_width, const int stride_h, const int stride_w, 
    const int window_h, const int window_w, Dtype* top_data) {

  CUDA_KERNEL_LOOP(index, nthreads) { //loop uber den input, jeder pixel
    /* (((height/stride_h) + (height%stride_h>0)) * ((width/stride_w) + (width%stride_w>0)),
     *   channels,
     *   window_h,
     *   window_w)
     */
    int c = index % channels;
    int n = index / channels;
    int origin_h = n / top_width;
    int origin_w = n % top_width;

    for (int h = -window_h/2; h < window_h/2; ++h)
      for (int w = -window_w/2; w < window_w/2; ++w) {
        int top_idx = index * window_h * window_w + //welches window
                  (h + window_h/2) * window_w + //welche zeile
                  (w + window_w/2); //welche spalte
        if (origin_h + h >= 0 && origin_h + h < bottom_height &&
            origin_w + w >= 0 && origin_w + w < bottom_width) {
          int bottom_idx =  c * bottom_height * bottom_width + //der richtige channel
                        (origin_h + h) * bottom_width + //richtige zeile
                        (origin_w + w);//richtige spalte
          top_data[top_idx] = bottom_data[bottom_idx];
        }
        else
          top_data[top_idx] = 0;
      }
  }
}


template <typename Dtype>
void SlidingWindowLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = top_height_ * top_width_ * channels_;
  SlidingWindowForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>> (
      count, bottom_data, channels_, bottom_height_, bottom_width_, top_height_, 
      top_width_, stride_h_, stride_w_, window_h_, window_w_, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void SlidingWindowBackward(const int nthreads, const Dtype* top_diff,
    const int channels, const int bottom_height, const int bottom_width, 
    const int top_height, const int top_width, const int stride_h, const int stride_w, 
    const int window_h, const int window_w, Dtype *bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    /* (((height/stride_h) + (height%stride_h>0)) * ((width/stride_w) + (width%stride_w>0)),
     *   channels,
     *   window_h,
     *   window_w)
     */
    int w = index % bottom_width;
    int h = (index / bottom_width) % bottom_height;
    int c = index / bottom_width / bottom_height;

    for (int off_h = -window_h + 1; off_h <= 0; ++off_h)
      for (int off_w = -window_w + 1; off_w <= 0; ++off_w) {
        int top_h = h + off_h;
        int top_w = w + off_w;
        if (top_h < -window_h/2 || top_w < -window_w/2 ||
            top_h >= top_height - window_h/2 || top_w >= top_width - window_w/2)
          continue;
        int top_n = (top_h + window_h/2) * top_width + (top_w + window_w/2);
        int top_idx = top_n * channels * window_h * window_w +
                      c * window_h * window_w +
                      (h - top_h) * window_w +
                      (w - top_w);
        bottom_diff[index] += top_diff[top_idx];
      }
  }
}

template <typename Dtype>
void SlidingWindowLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0])
    return;
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype *bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = channels_ * bottom_height_ * bottom_width_;
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  SlidingWindowBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_diff, channels_, bottom_height_, bottom_width_, top_height_,
      top_width_, stride_h_, stride_w_, window_h_, window_w_, bottom_diff);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(SlidingWindowLayer);


}
